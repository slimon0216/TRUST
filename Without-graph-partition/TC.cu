#include "hip/hip_runtime.h"
#include "TC.cuh"
#include "graph.h"
#include "herror.h"
#include "wtime.h"
#include <assert.h>
#include <bits/types/clock_t.h>
#include <cstdlib>
#include <iostream>
#include <iterator>
#include <math.h>
#include <queue>
#include <set>
#include <stdio.h>
#include <stdlib.h>

// #define dynamic
#define static
int BUCKET_SIZE = 100;
#define shared_BUCKET_SIZE 6
#define SUM_SIZE 1
#define USE_CTA 100
#define USE_WARP 2
#define without_combination 0
#define use_static 0

#define block_bucketnum 1024
#define warp_bucketnum 32

using namespace std;

__device__ void d_display(int *a, int column, int row, int start) {
  printf("\n");
  for (int i = 0; i < row; i++) {
    for (int j = 0; j < column; j++) {
      printf("%d\t", a[i * column + j + start]);
    }
    printf("\n");
  }
}

__device__ void kogge_sum(int *A, int *B, int len, int WARP_TID,
                          index_t *beg_pos) {
  /* We require enough threads for this method */
  int step = log2f(len) + 1;
  int i = WARP_TID;
  while (i < len) {
    A[i] = beg_pos[B[i] + 1] - beg_pos[B[i]];
    i += 32;
  }

  __syncwarp();
  for (i = 0; i < step; i++) {
    int pos = powf(2, i);
    int j = len - WARP_TID;
    while (j - pos >= 0) {
      int temp = A[j - pos];
      A[j] += temp;
      // printf("Write:%d , Read:%d , Written: %d\n",j,j-pos,A[j]);
      j -= 32;
    }
    // if(threadIdx.x==0){printf("\n\n");}
    __syncwarp();
  }
}

__device__ int linear_search(int neighbor, int *shared_partition,
                             int *partition, int *bin_count, int bin,
                             int BIN_START) {

  for (;;) {
    int i = bin;
    int len = bin_count[i];
    int step = 0;
    int nowlen;
    if (len < shared_BUCKET_SIZE)
      nowlen = len;
    else
      nowlen = shared_BUCKET_SIZE;
    while (step < nowlen) {
      if (shared_partition[i] == neighbor) {
        return 1;
      }
      i += block_bucketnum;
      step += 1;
    }

    len -= shared_BUCKET_SIZE;
    i = bin + BIN_START;
    step = 0;
    while (step < len) {
      if (partition[i] == neighbor) {
        return 1;
      }
      i += block_bucketnum;
      step += 1;
    }
    if (len + shared_BUCKET_SIZE < 99)
      break;
    bin++;
  }
  return 0;
}

__device__ int merge(int *A, int *B, int ai, int bi, int l1_e, int l2_e,
                     int steps) {
  /*Reminder: As the partition is coalesced, accessing next element in each
   * partition would require: next element --> prev + Warpsize */
  int WARPSIZE = 64;
  int count = 0;
  int steps_count = 0;
  while ((ai <= l1_e) && (bi <= l2_e)) {
    steps_count += 1;
    // printf("\nAI: %d, value: %d \t",ai, A[ai]);
    // printf("BI: %d, value: %d \t",bi, B[bi]);
    if (A[ai] > B[bi]) {
      bi += WARPSIZE;
    } else if (A[ai] < B[bi]) {
      ai += WARPSIZE;
    } else {
      count += 1;
      ai += WARPSIZE;
      bi += WARPSIZE;
    }
    // printf("\n");
    __syncthreads();
  }
  // printf("Thread: %d, count: %d \n",threadIdx.x,count);
  return count;
}

__device__ int device_binary_search(int *arr, int value, int start, int end) {
  int l = start, r = end;
  while (l < r - 1) {
    int mid = (l + r) >> 1;
    if (arr[mid] <= value)
      l = mid;
    else
      r = mid;
  }
  // if (arr[r]<=value) return r;
  if (arr[l] > value)
    return -1;
  return l;
}

int binary_search(int start, int end, int value, int *arr) {
  // printf("low:%d,high:%d,value:%f\n",start,end,value);
  int low = start;
  int high = end;
  int index = start;
  while (low <= high) {
    index = ((low + high) / 2);
    if (value < arr[index]) {
      // set high to index-1
      high = index - 1;
      // printf("high:%d\n",high);
    } else if (value > arr[index]) {
      // set low to index+1
      low = index + 1;
      // printf("low:%d\n",low);

    } else {
      break;
    }
  }
  // printf("Vaue: %d,Found: %d\n",value,arr[index]);
  return index;
}

int my_binary_search(int len, int val, index_t *beg) {
  int l = 0, r = len;
  while (l < r - 1) {
    int mid = (l + r) / 2;
    if (beg[mid + 1] - beg[mid] > val)
      l = mid;
    else
      r = mid;
  }
  if (beg[l + 1] - beg[l] <= val)
    return -1;
  return l;
}
__device__ int max_count(int *bin_count, int start, int end, int len) {
  int max_count = bin_count[start];
  int min_count = bin_count[start];
  int zero_count = 0;
  for (int i = start; i < end; i++) {
    if (bin_count[i] > max_count) {
      max_count = bin_count[i];
    }
    if (bin_count[i] < min_count) {
      min_count = bin_count[i];
    }
    if (bin_count[i] == 0) {
      zero_count += 1;
    }
  }
  // printf("%d,%d,%d\n",zero_count,max_count,len);
  return max_count - 1;
}

__global__ void dynamic_assign(vertex_t *adj_list, index_t *beg_pos,
                               int edge_count, int vertex_count, int *partition,
                               unsigned long long *GLOBAL_COUNT, int rank,
                               int total_process, int BUCKET_SIZE, int T_Group,
                               int *G_INDEX, int CHUNK_SIZE,
                               int warpfirstvertex, unsigned long long *gettime,
                               unsigned long long *maxcollision,
                               clock_t *thread_start, clock_t *thread_end,
                               clock_t *active_times) {

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  clock_t start_t, end_t, prev_t;
  active_times[tid] = 0;
  prev_t = start_t = clock(); // cycles


  // cache hash(i).len in the shared memory ?
  __shared__ int bin_count[block_bucketnum];
  __shared__ int shared_partition[block_bucketnum * shared_BUCKET_SIZE + 1];
  // __shared__ int shared_now,shared_workid;
  // __shared__ int useless[1024*9];
  // useless[threadIdx.x]=1;
  unsigned long long __shared__ G_counter;
  int WARPSIZE = 32;
  if (threadIdx.x == 0) {
    G_counter = 0;
  }
  // timetest
  // unsigned long long TT = 0, HT = 0, IT = 0;
  // unsigned long long __shared__ G_TT, G_HT, G_IT;
  // G_TT = 0, G_HT = 0, G_IT = 0;

  int BIN_START = blockIdx.x * block_bucketnum * BUCKET_SIZE;
  // __syncthreads();
  unsigned long long P_counter = 0;

  // unsigned long long start_time;

  // start_time = clock64();
  // CTA for large degree vertex
  int vertex = (blockIdx.x * total_process + rank) * CHUNK_SIZE;
  int vertex_end = vertex + CHUNK_SIZE;
  __shared__ int ver;
  while (vertex < warpfirstvertex) {
    // printf("%d,\t%d,\t%d\n", tid, vertex, warpfirstvertex);

    // int degree = beg_pos[vertex + 1] - beg_pos[vertex];
    // if (degree<=USE_CTA) break;
    int start = beg_pos[vertex];
    int end = beg_pos[vertex + 1];
    int now = threadIdx.x + start;
    int MODULO = block_bucketnum - 1;
    // int divide=(vert_count/blockDim.x);
    int BIN_OFFSET = 0;
    // clean bin_count
    for (int i = threadIdx.x; i < block_bucketnum; i += blockDim.x)
      bin_count[i] = 0;

    end_t = clock();
    active_times[tid] += end_t - prev_t;
    __syncthreads();
    prev_t = clock();

    // start_time = clock64();
    // count hash bin
    // build the hash table
    while (now < end) {
      int temp = adj_list[now];
      // bin is just the hash value
      int bin = temp & MODULO;
      int index;
      for (;;) {
        index = atomicAdd(&bin_count[bin], 1);
        if (index < shared_BUCKET_SIZE) {
          shared_partition[index * block_bucketnum + bin] = temp;
          break;
        } else if (index < BUCKET_SIZE) {
          index = index - shared_BUCKET_SIZE;
          partition[index * block_bucketnum + bin + BIN_START] = temp;
          break;
        }
        break;
        index = atomicAdd(&bin_count[bin], -1);
        bin = (bin + 1) % blockDim.x;
      }
      now += blockDim.x;
    }

    end_t = clock();
    active_times[tid] += end_t - prev_t;
    __syncthreads();
    prev_t = clock();

    // unsigned long long hash_time=clock64()-start_time;
    // start_time = clock64();
    // list intersection
    now = beg_pos[vertex];
    end = beg_pos[vertex + 1];
    if (without_combination) {
      while (now < end) {
        int neighbor = adj_list[now];
        int neighbor_start = beg_pos[neighbor];
        int neighbor_end = beg_pos[neighbor + 1];
        int neighbor_now = neighbor_start + threadIdx.x;
        while (neighbor_now < neighbor_end) {
          int temp = adj_list[neighbor_now];
          int bin = temp & MODULO;
          P_counter += linear_search(temp, shared_partition, partition,
                                     bin_count, bin + BIN_OFFSET, BIN_START);
          neighbor_now += blockDim.x;
        }
        now++;
      }
    } else {
      int superwarp_ID = threadIdx.x / 64;
      int superwarp_TID = threadIdx.x % 64;
      int workid = superwarp_TID;
      now = now + superwarp_ID;
      int neighbor = adj_list[now];
      int neighbor_start = beg_pos[neighbor];
      int neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
      while (now < end)
      // while (0)
      {
        while (now < end && workid >= neighbor_degree) {
          now += 16;
          workid -= neighbor_degree;
          neighbor = adj_list[now];
          neighbor_start = beg_pos[neighbor];
          neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
        }
        if (now < end) {
          int temp = adj_list[neighbor_start + workid];
          int bin = temp & MODULO;
          P_counter += linear_search(temp, shared_partition, partition,
                                     bin_count, bin + BIN_OFFSET, BIN_START);
        }
        // __syncthreads();
        workid += 64;
      }
    }
    if (0) {
      int workid = threadIdx.x;
      while (now < end)
      // while (0)
      {
        int neighbor = adj_list[now];
        int neighbor_start = beg_pos[neighbor];
        int neighbor_end = beg_pos[neighbor + 1];
        while (now < end && workid - (neighbor_end - neighbor_start) >= 0) {
          now++;
          workid -= (neighbor_end - neighbor_start);
          neighbor = adj_list[now];
          neighbor_start = beg_pos[neighbor];
          neighbor_end = beg_pos[neighbor + 1];
        }

        // if (threadIdx.x==0)
        // {
        // 	shared_now=now;
        // 	shared_workid=workid;
        // }
        // __syncthreads();
        if (now == end)
          break;
        int temp = adj_list[neighbor_start + workid];
        int bin = temp & MODULO;
        P_counter += linear_search(temp, shared_partition, partition, bin_count,
                                   bin + BIN_OFFSET, BIN_START);
        // __syncthreads();
        workid += blockDim.x;
        // workid=shared_workid+threadIdx.x+1;
        // now=shared_now;
      }
    }

    // unsigned long long intersection_time=clock64()-start_time;
    // if (threadIdx.x==0 &&degree>3000)
    // {
    // 	int max_len_collision= max_count(bin_count,0,blockDim.x,1);
    // 	printf("%d %d %d %d %lld
    // %lld\n",degree,vertex,blockIdx.x,max_len_collision,hash_time,intersection_time);
    // }

    end_t = clock();
    active_times[tid] += end_t - prev_t;
    __syncthreads();
    prev_t = clock();
    // if (vertex>1) break;
    if (use_static) {
      vertex += gridDim.x * total_process;
    } else {
      vertex++;
      if (vertex == vertex_end) {
        if (threadIdx.x == 0) {
          ver = atomicAdd(&G_INDEX[1], CHUNK_SIZE * total_process);
        }

        end_t = clock();
        active_times[tid] += end_t - prev_t;
        __syncthreads();
        prev_t = clock();
        vertex = ver;
        vertex_end = vertex + CHUNK_SIZE;
      }
    }
    // __syncthreads();
  }
  // __syncthreads();
  // unsigned long long CTA_time=clock64()-start_time;
  // start_time = clock64();

  // warp method
  int WARPID = threadIdx.x / WARPSIZE;
  int WARP_TID = threadIdx.x % WARPSIZE;
  int WARPDIM = blockDim.x * gridDim.x / WARPSIZE;
  vertex =
      warpfirstvertex +
      ((WARPID + blockIdx.x * blockDim.x / WARPSIZE) * total_process + rank) *
          CHUNK_SIZE;
  vertex_end = vertex + CHUNK_SIZE;
  while (vertex < vertex_count)
  // while (0)
  {
    // printf("%d,\t%d,\t%d\n", tid, vertex, vertex_count);
    unsigned long long start_time = clock64();
    int degree = beg_pos[vertex + 1] - beg_pos[vertex];
    if (degree < USE_WARP)
      break;
    int start = beg_pos[vertex];
    int end = beg_pos[vertex + 1];
    int now = WARP_TID + start;
    int MODULO = warp_bucketnum - 1;
    int BIN_OFFSET = WARPID * warp_bucketnum;
    // clean bin_count
    unsigned long long hash_start = clock64();

    for (int i = BIN_OFFSET + WARP_TID; i < BIN_OFFSET + warp_bucketnum;
         i += WARPSIZE)
      bin_count[i] = 0;
    // bin_count[threadIdx.x]=0;
    __syncwarp();

    // count hash bin
    while (now < end) {
      int temp = adj_list[now];
      int bin = temp & MODULO;
      bin += BIN_OFFSET;
      int index;
      for (;;) {
        index = atomicAdd(&bin_count[bin], 1);
        if (index < shared_BUCKET_SIZE) {
          shared_partition[index * block_bucketnum + bin] = temp;
          break;
        } else if (index < BUCKET_SIZE) {
          index = index - shared_BUCKET_SIZE;
          partition[index * block_bucketnum + bin + BIN_START] = temp;
          break;
        }
        break;
        index = atomicAdd(&bin_count[bin], -1);
        bin++;
        if (bin - BIN_OFFSET == 32)
          bin = BIN_OFFSET;
      }
      now += WARPSIZE;
    }
    __syncwarp();

    // unsigned long long hash_time=clock64()-hash_start;
    // unsigned long long intersection_start=clock64();
    // list intersection
    now = beg_pos[vertex];
    end = beg_pos[vertex + 1];

    if (without_combination) {
      while (now < end) {
        int neighbor = adj_list[now];
        int neighbor_start = beg_pos[neighbor];
        int neighbor_end = beg_pos[neighbor + 1];
        int neighbor_now = neighbor_start + WARP_TID;
        while (neighbor_now < neighbor_end) {
          int temp = adj_list[neighbor_now];
          int bin = temp & MODULO;
          P_counter += linear_search(temp, shared_partition, partition,
                                     bin_count, bin + BIN_OFFSET, BIN_START);
          neighbor_now += WARPSIZE;
        }
        now++;
      }
    } else {
      int workid = WARP_TID;
      while (now < end) {
        int neighbor = adj_list[now];
        int neighbor_start = beg_pos[neighbor];
        int neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
        // neighbor=__shfl_sync(0xffffffff,neighbor,31);
        // neighbor_start=__shfl_sync(0xffffffff,neighbor_start,31);
        // neighbor_degree=__shfl_sync(0xffffffff,neighbor_degree,31);

        while (now < end && workid >= neighbor_degree) {
          now++;
          workid -= neighbor_degree;
          neighbor = adj_list[now];
          neighbor_start = beg_pos[neighbor];
          neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
        }
        if (now < end) {
          int temp = adj_list[neighbor_start + workid];
          int bin = temp & MODULO;
          P_counter += linear_search(temp, shared_partition, partition,
                                     bin_count, bin + BIN_OFFSET, BIN_START);
        }
        __syncwarp();
        now = __shfl_sync(0xffffffff, now, 31);
        workid = __shfl_sync(0xffffffff, workid, 31);

        workid += WARP_TID + 1;

        // workid+=WARPSIZE;
      }
    }
    __syncwarp();
    // unsigned long long intersection_time=clock64()-intersection_start;
    // unsigned long long total_time=clock64()-start_time;
    // if(threadIdx.x%32==0){
    // 	// printf("%d %d %d\n",total_time, hash_time, intersection_time);
    // 	// TT+=total_time;
    // 	// HT+=hash_time;
    // 	// IT+=intersection_time;
    // 	gettime[vertex]=total_time;
    // 	maxcollision[vertex]=max_count(bin_count,BIN_OFFSET,BIN_OFFSET+WARPSIZE,0);
    // }
    // if(threadIdx.x%32==0){
    // 	gettime[vertex]=1;}
    __syncwarp();
    // if (vertex>1) break;
    if (use_static) {
      vertex += WARPDIM * total_process;
    } else {
      vertex++;
      if (vertex == vertex_end) {
        if (WARP_TID == 0) {
          vertex = atomicAdd(&G_INDEX[2], CHUNK_SIZE * total_process);
        }
        __syncwarp();
        vertex = __shfl_sync(0xffffffff, vertex, 0);
        vertex_end = vertex + CHUNK_SIZE;
      }
    }
  }

  // unsigned long long warp_time=clock64()-start_time;

  // if (threadIdx.x==0)
  // {
  // 	printf("%d %lld %lld\n",blockIdx.x,CTA_time,warp_time);
  // }
  atomicAdd(&G_counter, P_counter);
  // atomicAdd(&G_HT,HT);
  // atomicAdd(&G_TT,TT);
  // atomicAdd(&G_IT,IT);

  end_t = clock();
  active_times[tid] += end_t - prev_t;
  __syncthreads();
  if (threadIdx.x == 0) {
    // printf("%d\n",G_TT);
    atomicAdd(&GLOBAL_COUNT[0], G_counter);
    // atomicAdd(&GLOBAL_COUNT[1],G_TT);
    // atomicAdd(&GLOBAL_COUNT[2],G_HT);
    // atomicAdd(&GLOBAL_COUNT[3],G_IT);
  }
  end_t = clock();
  thread_start[tid] = start_t;
  thread_end[tid] = end_t;
}

struct arguments Triangle_count(int rank, char name[100], struct arguments args,
                                int total_process, int n_threads, int n_blocks,
                                int chunk_size) {

  // fprintf(stderr,"---------------Here----------------");
  int T_Group = 32;
  // int PER_BLOCK_WARP = n_threads / T_Group;
  int total = n_blocks * block_bucketnum * BUCKET_SIZE;
  unsigned long long *counter =
      (unsigned long long *)malloc(sizeof(unsigned long long) * 10);
  string json_file = name;
  graph *graph_d = new graph(json_file);

  // printf("Graph Adj Read: %d",graph_d->adj_list[10]);
  // int N_GPUS=argv[1];
  int deviceCount;
  HRR(hipGetDeviceCount(&deviceCount));
  // fprintf(stderr,"----------------Device count: %d\n",deviceCount);
  // HRR(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
  // HRR(hipFuncSetAttribute(reinterpret_cast<const void*>(dynamic_assign),hipFuncAttributePreferredSharedMemoryCarveout,16));
  // hipSetDevice();
  HRR(hipSetDevice((rank + 1) % deviceCount));
  hipDeviceProp_t devProp;
  HRR(hipGetDeviceProperties(&devProp, rank));
  index_t vertex_count = graph_d->vert_count;
  index_t edge_count = graph_d->edge_count;
  index_t edge_list_count = graph_d->edge_list_count;
  index_t edges = graph_d->edge_count;
  /* Preprocessing Step to calculate the ratio */
  int *prefix = (int *)malloc(sizeof(int) * vertex_count);

  // USE_CTA = 100, if degree(v) > 100, use an entire block
  // find the first vertex not using a block
  int warpfirstvertex =
      my_binary_search(vertex_count, USE_CTA, graph_d->beg_pos) + 1;

  // cout<<my_binary_search(vertex_count,1,graph_d->beg_pos)<<'
  // '<<my_binary_search(vertex_count,0,graph_d->beg_pos)<<endl; printf("divide
  // position%d %d %d
  // %d\n",graph_d->beg_pos[warpfirstvertex+1]-graph_d->beg_pos[warpfirstvertex],warpfirstvertex,graph_d->beg_pos[warpfirstvertex]-graph_d->beg_pos[warpfirstvertex-1],graph_d->beg_pos[warpfirstvertex-1]-graph_d->beg_pos[warpfirstvertex-2]);

  // fprintf(stderr,"Rank: %d, Devicecount: %d,  Start: %d, End: %d, Selected:
  // %d\n",rank,deviceCount,E_START,E_END,(rank%deviceCount));

  // cout<<edge_count<<' '<<rank<<' '<<total_process<<endl;
  // for (int i=graph_d->beg_pos[9631];i<graph_d->beg_pos[9632];i++)
  // {
  // 	cout<<graph_d->adj_list[i]<<endl;
  // }
  // cout<<vertex_count<<' '<<edge_count<<endl;
  // cout<<graph_d->beg_pos[vertex_count-100]<<'
  // '<<graph_d->beg_pos[vertex_count-200]<<endl;

  int *BIN_MEM;
  unsigned long long *GLOBAL_COUNT, *g_gettime, *g_maxcollision;
  int *G_INDEX;
  index_t *d_beg_pos;
  vertex_t *d_adj_list;
  float memory_req =
      (sizeof(int) * total + sizeof(index_t) * (vertex_count + 1) +
       sizeof(vertex_t) * (edge_count) + sizeof(vertex_t) * (edge_list_count)) /
      (1024 * 1024);
  // fprintf(stderr,"-------------------GPU: %d, Memory required: %f
  // MB\n",rank,memory_req);
  // printf("%f\n",memory_req);
  HRR(hipMalloc((void **)&GLOBAL_COUNT, sizeof(unsigned long long) * 10));
  HRR(hipMalloc((void **)&g_gettime,
                 sizeof(unsigned long long) * (vertex_count + 1)));
  HRR(hipMalloc((void **)&g_maxcollision,
                 sizeof(unsigned long long) * (vertex_count + 1)));
  HRR(hipMalloc((void **)&G_INDEX, sizeof(int) * 3));
  HRR(hipMalloc((void **)&d_beg_pos, sizeof(index_t) * (vertex_count + 1)));
  HRR(hipMalloc((void **)&d_adj_list, sizeof(vertex_t) * (edge_count)));
  // HRR(hipMalloc((void **) &d_edge_list,sizeof(vertex_t)*(vertex_count+1)));
  // Swap edge list count with Eend - estart; --> gives error; may add some more

  // fprintf(stderr,">>>>>>>>>>>>>>>>>Malloc:adj_list[10]:
  // %d\n",graph_d->adj_list[10]);

  int nowindex[3];
  nowindex[0] = chunk_size * n_blocks * n_threads / T_Group;
  nowindex[1] = chunk_size * (n_blocks * total_process + rank);
  nowindex[2] =
      warpfirstvertex +
      chunk_size * (n_blocks * n_threads / T_Group * total_process + rank);
  // unsigned long long cou=0;
  // int nowindex=0;

  HRR(hipMemcpy(G_INDEX, &nowindex, sizeof(int) * 3, hipMemcpyHostToDevice));
  // HRR(hipMemcpy(GLOBAL_COUNT, &cou, sizeof(unsigned long long),
  // hipMemcpyHostToDevice));
  // HRR(hipMemcpy(d_edge_list,graph_d->edge_list,sizeof(vertex_t)*(vertex_count+1),
  // hipMemcpyHostToDevice));
  // for (int i = 1; i < vertex_count+1; ++i) {
  //   cout << graph_d->beg_pos[i] - graph_d->beg_pos[i-1] << ' ';
  // }
  // exit(1);
  HRR(hipMemcpy(d_beg_pos, graph_d->beg_pos,
                 sizeof(index_t) * (vertex_count + 1), hipMemcpyHostToDevice));
  HRR(hipMemcpy(d_adj_list, graph_d->adj_list, sizeof(vertex_t) * edge_count,
                 hipMemcpyHostToDevice));
  // fprintf(stderr,">>>>>>>>>>>>>>>>>>>Memcopy completed");
  // HRR(hipDeviceSetLimit(hipLimitMallocHeapSize, 128*1024*1024));
  double t1 = wtime();
  double cmp_time;
  HRR(hipMalloc((void **)&BIN_MEM, sizeof(int) * total));

  clock_t *thread_start;
  clock_t *thread_end;
  clock_t *active_times;
  HRR(hipMallocManaged(&thread_start, sizeof(clock_t) * n_blocks * n_threads));
  HRR(hipMallocManaged(&thread_end, sizeof(clock_t) * n_blocks * n_threads));
  HRR(hipMallocManaged(&active_times, sizeof(clock_t) * n_blocks * n_threads));

  if (1) {
    double time_start = clock();
    // HRR(hipMalloc((void **) &BIN_MEM,sizeof(int)*total));
    dynamic_assign<<<n_blocks, n_threads>>>(
        d_adj_list, d_beg_pos, edge_count, vertex_count, BIN_MEM, GLOBAL_COUNT,
        rank, total_process, BUCKET_SIZE, T_Group, G_INDEX, chunk_size,
        warpfirstvertex, g_gettime, g_maxcollision, thread_start, thread_end,
        active_times);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      printf("Error: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    HRR(hipDeviceSynchronize());
    // HRR(hipFree(BIN_MEM));
    cmp_time = clock() - time_start;
    cmp_time = cmp_time / CLOCKS_PER_SEC;
  }
  HRR(hipFree(BIN_MEM));

  HRR(hipMemcpy(counter, GLOBAL_COUNT, sizeof(unsigned long long) * 10,
                 hipMemcpyDeviceToHost));
  // unsigned long long *gettime= new unsigned long long[vertex_count+1];
  // HRR(hipMemcpy(gettime,g_gettime,sizeof(unsigned long
  // long)*(vertex_count+1), hipMemcpyDeviceToHost)); unsigned long long
  // *maxcollision= new unsigned long long[vertex_count+1];
  // HRR(hipMemcpy(maxcollision,g_maxcollision,sizeof(unsigned long
  // long)*(vertex_count+1), hipMemcpyDeviceToHost));

  HRR(hipFree(GLOBAL_COUNT));
  HRR(hipFree(g_gettime));
  HRR(hipFree(g_maxcollision));
  HRR(hipFree(G_INDEX));
  HRR(hipFree(d_beg_pos));
  HRR(hipFree(d_adj_list));
  // HRR(hipFree(d_edge_list));
  // free(counter);
  free(prefix);
  delete graph_d;
  args.time = cmp_time;
  args.count = counter[0];
  // cout<<counter[0]<<endl;
  // printf("%lld\n",args.count);
  // cout<<counter[1]<<' '<<counter[2]<<' '<<counter[3]<<endl;
  // for (int i=0;i<vertex_count;i++)
  // 	if (gettime[i]>0)
  // 		cout<<gettime[i]<<'
  // '<<graph_d->beg_pos[i+1]-graph_d->beg_pos[i]<<'
  // '<<maxcollision[i]+1<<endl;

  // for (int i=0;i<vertex_count;i++)
  // 	if (gettime[i]==0&&i%total_process==rank ||
  // gettime[i]==1&&i%total_process!=rank)
  // 	{
  // 		cout<<i<<endl;
  // 		break;
  // 	}

  string log_name = string(name);
  log_name = string("log_") + basename(log_name.substr(0, log_name.length()-1).c_str());
  // cout <<log_name<<endl;
  log_name += "_" + to_string(n_blocks) + 'x' + to_string(n_threads);
  FILE *log_file = fopen(log_name.c_str(), "w"); 
  printf("clock rate: %dKHz\n", devProp.clockRate);
  fprintf(log_file, "%f\n", cmp_time*1000);
  fprintf(log_file, "thread_id,start_time,end_time,clock_time,active_time\n");
  for (int i = 0; i < n_blocks * n_threads; ++i) {
    fprintf(log_file, "%d,%ld,%ld,%ld,%ld\n", i, thread_start[i], thread_end[i],
            thread_end[i] - thread_start[i], active_times[i]);
  }
  fclose(log_file);
  args.edge_count = edges;
  args.degree = edges / vertex_count;
  args.vertices = vertex_count;
  return args;
}
