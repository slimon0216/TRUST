#include "hip/hip_runtime.h"
#include "TC.cuh"
#include "comm.cuh"
#include "graph.h"
#include "herror.h"
#include "wtime.h"
#include <assert.h>
#include <cstring>
#include <fstream>
#include <iostream>
#include <iterator>
#include <math.h>
#include <queue>
#include <set>
#include <sstream>
#include <stdio.h>
#include <stdlib.h>
#include <string>

// #define dynamic
#define static
int BUCKET_SIZE = 100;
#define shared_BUCKET_SIZE 6
#define SUM_SIZE 1
#define USE_CTA 100
#define USE_WARP 2
#define without_combination 0
#define use_static 0

using namespace std;

__device__ void d_display(int *a, int column, int row, int start) {
  printf("\n");
  for (int i = 0; i < row; i++) {
    for (int j = 0; j < column; j++) {
      printf("%d\t", a[i * column + j + start]);
    }
    printf("\n");
  }
}

__device__ void kogge_sum(int *A, int *B, int len, int WARP_TID,
                          index_t *beg_pos) {
  /* We require enough threads for this method */
  int step = log2f(len) + 1;
  int i = WARP_TID;
  while (i < len) {
    A[i] = beg_pos[B[i] + 1] - beg_pos[B[i]];
    i += 32;
  }

  __syncwarp();
  for (i = 0; i < step; i++) {
    int pos = powf(2, i);
    int j = len - WARP_TID;
    while (j - pos >= 0) {
      int temp = A[j - pos];
      A[j] += temp;
      // printf("Write:%d , Read:%d , Written: %d\n",j,j-pos,A[j]);
      j -= 32;
    }
    // if(threadIdx.x==0){printf("\n\n");}
    __syncwarp();
  }
}

__device__ int linear_search(int neighbor, int *shared_partition,
                             int *partition, int *bin_count, int bin,
                             int BIN_START) {

  for (;;) {
    int i = bin;
    int len = bin_count[i];
    int step = 0;
    int nowlen;
    if (len < shared_BUCKET_SIZE)
      nowlen = len;
    else
      nowlen = shared_BUCKET_SIZE;
    while (step < nowlen) {
      if (shared_partition[i] == neighbor) {
        return 1;
      }
      i += blockDim.x;
      step += 1;
    }

    len -= shared_BUCKET_SIZE;
    i = bin + BIN_START;
    step = 0;
    while (step < len) {
      if (partition[i] == neighbor) {
        return 1;
      }
      i += blockDim.x;
      step += 1;
    }
    if (len + shared_BUCKET_SIZE < 99)
      break;
    bin++;
  }
  return 0;
}
__device__ int merge(int *A, int *B, int ai, int bi, int l1_e, int l2_e,
                     int steps) {
  /*Reminder: As the partition is coalesced, accessing next element in each
   * partition would require: next element --> prev + Warpsize */
  int WARPSIZE = 64;
  int count = 0;
  int steps_count = 0;
  while ((ai <= l1_e) && (bi <= l2_e)) {
    steps_count += 1;
    // printf("\nAI: %d, value: %d \t",ai, A[ai]);
    // printf("BI: %d, value: %d \t",bi, B[bi]);
    if (A[ai] > B[bi]) {
      bi += WARPSIZE;
    } else if (A[ai] < B[bi]) {
      ai += WARPSIZE;
    } else {
      count += 1;
      ai += WARPSIZE;
      bi += WARPSIZE;
    }
    // printf("\n");
    __syncthreads();
  }
  // printf("Thread: %d, count: %d \n",threadIdx.x,count);
  return count;
}

__device__ int device_binary_search(int *arr, int value, int start, int end) {
  int l = start, r = end;
  while (l < r - 1) {
    int mid = (l + r) >> 1;
    if (arr[mid] <= value)
      l = mid;
    else
      r = mid;
  }
  // if (arr[r]<=value) return r;
  if (arr[l] > value)
    return -1;
  return l;
}

int binary_search(int start, int end, int value, int *arr) {
  // printf("low:%d,high:%d,value:%f\n",start,end,value);
  int low = start;
  int high = end;
  int index = start;
  while (low <= high) {
    index = ((low + high) / 2);
    if (value < arr[index]) {
      // set high to index-1
      high = index - 1;
      // printf("high:%d\n",high);
    } else if (value > arr[index]) {
      // set low to index+1
      low = index + 1;
      // printf("low:%d\n",low);

    } else {
      break;
    }
  }
  // printf("Vaue: %d,Found: %d\n",value,arr[index]);
  return index;
}

int my_binary_search(int len, int val, index_t *beg) {
  int l = 0, r = len;
  while (l < r - 1) {
    int mid = (l + r) / 2;
    if (beg[mid + 1] - beg[mid] > val)
      l = mid;
    else
      r = mid;
  }
  if (beg[l + 1] - beg[l] <= val)
    return -1;
  return l;
}
__device__ int max_count(int *bin_count, int start, int end, int len) {
  int max_count = bin_count[start];
  int min_count = bin_count[start];
  int zero_count = 0;
  for (int i = start; i < end; i++) {
    if (bin_count[i] > max_count) {
      max_count = bin_count[i];
    }
    if (bin_count[i] < min_count) {
      min_count = bin_count[i];
    }
    if (bin_count[i] == 0) {
      zero_count += 1;
    }
  }
  // printf("%d,%d,%d\n",zero_count,max_count,len);
  return max_count - 1;
}

// void graph_reordering(graph *graph_temp)
// {

// }

// __global__ void
// warp_hash_count(vertex_t* adj_list, index_t* beg_pos, vertex_t* edge_list,
// int edge_count, int vertex_count,int edge_list_count, int *partition,unsigned
// long long *GLOBAL_COUNT,int E_START, int E_END, int device, int BUCKETS, int
// G_BUCKET_SIZE, int T_Group, int *G_INDEX, int CHUNK_SIZE)
// {
// 	// Uncomment the lines below and change partition to Gpartition for
// using shared version 	int *part; 	int
// tid=threadIdx.x+blockIdx.x*blockDim.x; 	int WARPSIZE=T_Group; 	int
// PER_BLOCK_WARP=blockDim.x/WARPSIZE; 	int G_WARPID= tid/WARPSIZE; 	int
// WARPID =
// threadIdx.x/WARPSIZE; 	int WARP_TID=threadIdx.x%WARPSIZE; 	int
// __shared__ bin_count[32*4];
// 	//int __shared__ partition[160*4];
// 	int o=0, i =0;
// 	unsigned long long __shared__ G_counter;
// 	// int __shared__ warp_counter[4];
// 	// int __shared__ index[4];
// 	if (threadIdx.x==0)
// 	{
// 		G_counter=0;
// 	}
// 	unsigned long long P_counter=0;
// 	int BINsize = BUCKETS*G_BUCKET_SIZE;
// 	//int BINsize = BUCKETS*5;
// 	int BIN_START = G_WARPID*BINsize;
// 	//int BIN_START = WARPID*BINsize;
// 	int total_WARPS=gridDim.x* blockDim.x/32;
// 	int STOP = E_END - total_WARPS;
// 	//int i=G_WARPID+E_START;
// 	int RANGE= E_END-E_START;
// 	int BIN_OFFSET= WARPID*BUCKETS;

// 	int MODULO=BUCKETS-1;
// 	// int DIVIDE=(vertex_count+BUCKETS)/BUCKETS;

// 	double start_time;

// 	int SubSize=4;
// 	int SubNumber=WARPSIZE/SubSize; // Total number of sub groups in warp
// 	int Sub_Tid= WARP_TID%SubSize;	// Thread ID within a warp
// 	int SUB_ID = WARP_TID / SubSize;// ID of sub threads
// 	__shared__ int prefix_sum[SUM_SIZE*4];
// 	// #pragma unroll 5

// 	int START_INDEX= 0;
// 	//E_END=200;

// #ifdef dynamic
// 	while (*G_INDEX<E_END)
// #endif
// 	{
// #ifdef dynamic

// 		if(WARP_TID==0)
// 		{
// 			if (START_INDEX==0)
// 			{
// 				START_INDEX=(blockIdx.x*4+WARPID)*CHUNK_SIZE;
// 			}
// 			else
// 			{
// 				START_INDEX= atomicAdd(G_INDEX, CHUNK_SIZE);
// 			}
// 		}
// 		__syncwarp();
// 		START_INDEX= __shfl_sync(0xffffffff,START_INDEX,0);
// 		int i_end= START_INDEX + CHUNK_SIZE;
// 		if (i_end>E_END) i_end=E_END;
// #endif

// #ifdef static
// 		int i_end=E_END;
// 		START_INDEX=blockIdx.x*4+WARPID;
// #endif
// 		while((START_INDEX < i_end))
// 		{

// 			// if (WARP_TID==0)
// 			// 	printf("%d\n",START_INDEX);
// 			// N2 is for hashing and N1 is lookup
// 			//if(threadIdx.x==0){printf("I: %d,G:
// %d\n",i,G_WARPID);}

// 			int N2_start=beg_pos[START_INDEX];
// 			int N2_end= beg_pos[START_INDEX+1];
// 			int L2= N2_end-N2_start;

// 			//-------------------- If L2 is equal to 0,
// continue------------------- 			if(L2<=1)
// 			{
// #ifdef dynamic
// 				START_INDEX+=1;
// #endif
// #ifdef static
// 				START_INDEX+=gridDim.x*PER_BLOCK_WARP;
// #endif
// 				continue;
// 			}
// 			//------------------------Clear bin
// counter--------------------------------
// 			//if(WARP_TID==0){printf("\n%d,%d,",START_INDEX,L2);}
// 			int id=WARP_TID+BIN_OFFSET;
// 			int end = BIN_OFFSET+BUCKETS;
// 			while(id<(end))
// 			{
// 				bin_count[id]=0;
// 				//printf("BIN: %d\n",id);
// 				id+=WARPSIZE;
// 			}
// 			__syncwarp();
// 			//--------------------------Hash source
// node------------------------------ 			start_time = clock64();
// int start=WARP_TID
// + N2_start;
// 			// BIN_OFFSET is for count of number of element of each
// bin for all 4 warps
// 			// Hash one list
// 			while(start<N2_end)
// 			{
// 				//if(threadIdx.x==0){printf("I: %d, Hashing:
// %d\n",i,L2);} 				int temp= adj_list[start];
// int bin=temp & MODULO; 				int
// index=atomicAdd(&bin_count[bin+BIN_OFFSET],1);
// partition[index*BUCKETS+ bin + BIN_START]=temp;
// 				//{printf("thread: %d,warp:%d, write: %d bin %d,
// index %d at:
// %d\n",threadIdx.x,WARPID,temp,bin,index,(index*WARPSIZE+bin+BIN_START));}
// 				start+=WARPSIZE;
// 			}
// 			__syncwarp();
// 			double hash_time=clock64();
// 			hash_time-=start_time;
// 			double prefix_sum_time;
// 			double with_combination_time;
// 			double without_combination_time;

// 			//for large degree neighbor
// 			if (0)
// 			{
// 				int N_start=N2_start;
// 				// printf("%d
// %d\n",edge_list[START_INDEX],START_INDEX); 				while
// (N_start<N2_start+L2)
// 				{
// 					int v=adj_list[N_start];
// 					int s=beg_pos[v]+WARP_TID;
// 					int e=beg_pos[v+1];
// 					while (s<e)
// 					{
// 						int neighbor=adj_list[s];
// 						int bin=neighbor & MODULO;
// 						P_counter+=linear_search(neighbor,partition,bin_count,bin,BIN_OFFSET,BIN_START,BUCKETS);
// 						s+=32;
// 					}
// 					N_start++;
// 				}

// 				N2_start=N2_start+L2;
// 				L2=N2_end-N2_start;
// 				if (L2<0)
// printf("%d\n",beg_pos[START_INDEX+1]-beg_pos[START_INDEX],edge_list[START_INDEX]);
// 			}
// 			// for short degree neighbor
// 			if (L2>0)
// 			{
// 				if (L2<SUM_SIZE)
// 				{
// 					__syncwarp();

// 					start_time = clock64();
// 					kogge_sum(&prefix_sum[WARPID*SUM_SIZE],&adj_list[N2_start],L2,WARP_TID,beg_pos);

// 					__syncwarp();

// 					prefix_sum_time=clock64();
// 					prefix_sum_time-=start_time;
// 					start_time = clock64();

// 					int N_start=WARP_TID;
// 					while
// (N_start<prefix_sum[WARPID*SUM_SIZE+L2-1])
// 					{
// 						int
// v=device_binary_search(prefix_sum,N_start,WARPID*SUM_SIZE,WARPID*SUM_SIZE+L2-1);
// 						int u=N_start;
// 						if (v>=0)
// 						{
// 							u-=prefix_sum[v];
// 							v-=WARPID*SUM_SIZE;
// 						}
// 						v++;
// 						v=adj_list[N2_start+v];
// 						int
// neighbor=adj_list[beg_pos[v]+u];

// 						// printf("%d %d
// %d\n",u,v,neighbor); 						int bin=neighbor
// & MODULO;
// 						P_counter+=linear_search(neighbor,partition,bin_count,bin,BIN_OFFSET,BIN_START,BUCKETS);
// 						N_start+=32;
// 					}
// 					__syncwarp();

// 					with_combination_time=clock64();
// 					with_combination_time-=start_time;
// 					start_time = clock64();
// 				}
// 				else
// 				{
// 					int Nstart= N2_start+ SUB_ID;
// 					while (Nstart<N2_end)
// 					{
// 						int neighbor= adj_list[Nstart];
// 						////if(threadIdx.x==0){printf("Intersecting:
// %d\n",neighbor);} 						int
// N_start=beg_pos[neighbor]; 						int
// N_end= beg_pos[neighbor+1]; 						int L1=
// N_end-N_start;
// 						//if(WARP_TID==0){printf("%d,",L1);}
// 						start=Sub_Tid + N_start;
// 						while(start<N_end)
// 						{
// 							int
// neighbor=adj_list[start]; 							int
// bin=neighbor & MODULO;
// 							P_counter+=linear_search(neighbor,partition,bin_count,bin,BIN_OFFSET,BIN_START,BUCKETS);
// 							start+=SubSize;
// 							//printf("Tid: %d,
// Search:%d\n",threadIdx.x,neighbor);
// 						}
// 						Nstart+=SubNumber;
// 					}

// 					__syncwarp();
// 					without_combination_time=clock64();
// 					without_combination_time-=start_time;
// 				}
// 			}
// 			// if (WARP_TID==0)
// 			//
// printf("%d,%d,%d,%lf,%lf,%lf,%lf\n",START_INDEX,L2,prefix_sum[WARPID*SUM_SIZE+L2-1],hash_time,prefix_sum_time,with_combination_time,without_combination_time);
// 			// unsigned long long int stop_time= clock64();
// 			// unsigned long long int search_time =
// (stop_time-start_time);
// 			//if(WARP_TID==0){printf("%.d\n",search_time);}

// #ifdef dynamic
// 			START_INDEX+=1;
// #endif
// #ifdef static
// 			START_INDEX+=gridDim.x*PER_BLOCK_WARP;
// #endif
// 			// if (WARP_TID==0)
// 			// 	printf("startindex%d\n",START_INDEX);
// 			__syncwarp();
// 		}
// 	}
// 	//unsigned long long int stop_time_warp=clock64();
// 	//unsigned long long int warp_time= stop_time_warp -start_time_warp;
// 	// if(WARP_TID==0)
// 	// 	{
// 	// 		printf("%d, %d\n",G_WARPID,warp_time);
// 	// 	}
// 	atomicAdd(&G_counter,P_counter);
// 	__syncthreads();
// 	if(threadIdx.x==0)
// 	{
// 		atomicAdd(&GLOBAL_COUNT[0],G_counter);
// 		// printf("%lld\n",GLOBAL_COUNT[0]);
// 	}

// 	//if(tid==0){printf("Device: %d, Count:%d\n",device,GLOBAL_COUNT[0]);}
// }

// __global__ void
// CTA_hash_count(vertex_t* adj_list, index_t* beg_pos, vertex_t* edge_list, int
// edge_count, int vertex_count,int edge_list_count, int *partition,unsigned
// long long *GLOBAL_COUNT,int E_START, int E_END, int device, int BUCKETS, int
// G_BUCKET_SIZE,int T_Group)
// {
// 	// Uncomment the lines below and change partition to Gpartition for
// using shared version 	int *part; 	int S_BUCKET_SIZE=320; 	int
// tid=threadIdx.x+blockIdx.x*blockDim.x; 	int WARPSIZE=blockDim.x;
// int PER_BLOCK_WARP=1; 	int G_WARPID= blockIdx.x; 	int WARPID = 0;
// int WARP_TID=threadIdx.x; 	int __shared__ bin_count[256*4];
// 	//int __shared__ partition[160*4];
// 	int __shared__ G_counter;
// 	int __shared__ warp_counter[4];
// 	int __shared__ index[4];
// 	G_counter=0;
// 	int P_counter=0;
// 	int BINsize = BUCKETS*G_BUCKET_SIZE;
// 	//int BINsize = BUCKETS*5;
// 	int BIN_START = G_WARPID*BINsize;
// 	//int BIN_START = WARPID*BINsize;

// 	// if (WARP_TID==0)
// 	// {
// 	// 	index[WARPID]= atomicAdd(&G_INDEX[0], 1);
// 	// }
// 	// __syncwarp();
// 	//int i=index[WARPID];
// 	//int i=0,o=0;
// 	int total_WARPS=gridDim.x* blockDim.x/32;
// 	int i=G_WARPID+E_START;
// 	int RANGE= E_END-E_START;
// 	int BIN_OFFSET= WARPID*BUCKETS;
// 	int count;
// 	int MODULO=BUCKETS-1;
// 	//TODO: Static assignment to dynamic assignment of vertices
// 	#pragma unroll 5
// 	//if(threadIdx.x==0){printf("I: %d\n",RANGE);}
// 	clock_t start_time_warp=clock();
// 	while(i<( E_END))
// 	{
// 		// N2 is for hashing and N1 is lookup
// 		//if(threadIdx.x==0){printf("I: %d\n",i);}
// 		int N2_start=beg_pos[i];
// 		int N2_end= beg_pos[i+1];
// 		int L2= N2_end-N2_start;
// 		//-------------------- If L2 is equal to 0,
// continue------------------- 		if(L2==0 || L2==1)
// 		{
// 			i+=gridDim.x*PER_BLOCK_WARP;

// 			continue;
// 			}
// 		//------------------------Clear bin
// counter-------------------------------- 		int
// id=WARP_TID+BIN_OFFSET; int end = BIN_OFFSET+BUCKETS;
// while(id<(end))
// 		{
// 			bin_count[id]=0;
// 			id+=WARPSIZE;
// 		}
// 		__syncthreads();
// 		//--------------------------Hash source
// node------------------------------
// 		// clock_t start_time = clock();
// 		int start=WARP_TID + N2_start;
// 		// BIN_OFFSET is for count of number of element of each bin for
// all 4 warps
// 		// Hash one list
// 		while(start<N2_end)
// 		{
// 			//if(threadIdx.x==0){printf("I: %d, Hashing:
// %d\n",i,L2);} 			int temp= adj_list[start];
// int bin=temp & MODULO; 			int
// index=atomicAdd(&bin_count[bin+BIN_OFFSET],1);
// partition[index*BUCKETS+ bin + BIN_START]=temp;
// 			//{printf("thread: %d,warp:%d, write: %d bin %d, index
// %d at:
// %d\n",threadIdx.x,WARPID,temp,bin,index,(index*WARPSIZE+bin+BIN_START));}
// 			start+=WARPSIZE;
// 		}

// 		__syncthreads();
// 		// clock_t stop_time = clock();
// 		// float hash_time = float(stop_time-start_time);
// 		// int max_len_collision=
// max_count(bin_count,BIN_OFFSET,BIN_OFFSET+BUCKETS,L2);
// 		//--------------------------Loop over the neighbors of the
// vertex-------------------------------- 		start_time=clock();
// int Nstart= N2_start; 		while (Nstart<N2_end)
// 		{
// 			int neighbor= adj_list[Nstart];
// 			////if(threadIdx.x==0){printf("Intersecting:
// %d\n",neighbor);} 			int N_start=beg_pos[neighbor];
// int N_end= beg_pos[neighbor+1];
// 			// int L1= N_end-N_start;
// 			//-----------------------------
// 			// if(L1==0)
// 			// {
// 			// 	Nstart+=1;
// 			// 	continue;
// 			// }
// 			//---------------------------
// 			start=WARP_TID + N_start;
// 			while(start<N_end)
// 			{
// 				count=0;
// 				int neighbor=adj_list[start];
// 				int bin=neighbor&MODULO;
// 				count=linear_search(neighbor,partition,bin_count,bin,BIN_OFFSET,BIN_START,BUCKETS);
// 				P_counter+=count;
// 				start+=WARPSIZE;
// 				//printf("Tid: %d,
// Search:%d\n",threadIdx.x,neighbor);
// 			}
// 			Nstart+=1;

// 			__syncthreads();
// 		}
// 		stop_time=clock();
// 		float search_time= float(stop_time-start_time);

// 		// if(WARP_TID==0)
// 		// {
// 		// 	printf("%d, %d, %d,%.2f, %.2f\n",i,L2,max_len_collision
// ,hash_time, search_time);
// 		// }
// 		i+=gridDim.x*PER_BLOCK_WARP;
// 	}
// 	clock_t stop_time_warp=clock();
// 	float warp_time= float(stop_time_warp -start_time_warp);
// 	if(WARP_TID==0)
// 		{
// 			//printf("%d, %.2f\n",G_WARPID,warp_time);
// 		}
// 	atomicAdd(&G_counter,P_counter);
// 	__syncthreads();
// 	if(threadIdx.x==0){atomicAdd(&GLOBAL_COUNT[0],G_counter);}

// 	//if(threadIdx.x==0){printf("Device: %d,
// Count:%d\n",device,GLOBAL_COUNT[0]);}
// }

__global__ void
dynamic_assign(vertex_t *adj_list_HT, index_t *beg_pos_HT,
               vertex_t *adj_list_intersection, index_t *beg_pos_intersection,
               vertex_t *adj_list_neighbor, index_t *beg_pos_neighbor,
               int edge_count, int vertex_count, int edge_list_count,
               int *partition, unsigned long long *GLOBAL_COUNT, int rank,
               int total_process, int BUCKET_SIZE, int T_Group, int *G_INDEX,
               int CHUNK_SIZE, int warpfirstvertex, int nothreadfirstvertex,
               int *vertexmap, unsigned long long *gettime,
               unsigned long long *maxcollision) {

  // printf("thread%d",threadIdx.x);
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  __shared__ int bin_count[1024];
  __shared__ int shared_partition[1024 * shared_BUCKET_SIZE + 1];
  unsigned long long __shared__ G_counter;
  int WARPSIZE = 32;
  if (threadIdx.x == 0) {
    G_counter = 0;
  }
  // timetest
  // unsigned long long TT=0,HT=0,IT=0;
  // unsigned long long __shared__ G_TT,G_HT,G_IT;
  // G_TT=0,G_HT=0,G_IT=0;

  __syncthreads();
  unsigned long long P_counter = 0;

  // unsigned long long start_time;

  // start_time = clock64();
  // CTA for large degree vertex
  int vertex = (blockIdx.x * total_process + rank) * CHUNK_SIZE;
  int vertex_end = vertex + CHUNK_SIZE;
  __shared__ int ver;
  while (vertex < warpfirstvertex) {
    int vertexID = vertexmap[vertex];
    int BINsize = blockDim.x * BUCKET_SIZE;
    int BIN_START = blockIdx.x * BINsize;
    // if (degree<=USE_CTA) break;
    int start = beg_pos_HT[vertexID];
    int end = beg_pos_HT[vertexID + 1];
    int degree = end - start;
    int now = threadIdx.x + start;
    int MODULO = blockDim.x - 1;
    // int divide=(vert_count/blockDim.x);
    int BIN_OFFSET = 0;
    // clean bin_count
    bin_count[threadIdx.x] = 0;
    __syncthreads();

    // start_time = clock64();
    // count hash bin
    while (now < end) {
      int temp = adj_list_HT[now];
      int bin = temp & MODULO;
      int index;
      for (;;) {
        index = atomicAdd(&bin_count[bin], 1);
        if (index < shared_BUCKET_SIZE) {
          shared_partition[index * blockDim.x + bin] = temp;
          break;
        } else if (index < BUCKET_SIZE) {
          index = index - shared_BUCKET_SIZE;
          partition[index * blockDim.x + bin + BIN_START] = temp;
          break;
        }
        break;
        index = atomicAdd(&bin_count[bin], -1);
        bin = (bin + 1) % blockDim.x;
      }
      now += blockDim.x;
    }
    __syncthreads();

    // unsigned long long hash_time=clock64()-start_time;
    // start_time = clock64();
    // list intersection
    now = beg_pos_intersection[vertexID];
    end = beg_pos_intersection[vertexID + 1];
    if (without_combination) {
      while (now < end) {
        int neighbor = adj_list_intersection[now];
        int neighbor_start = beg_pos_neighbor[neighbor];
        int neighbor_end = beg_pos_neighbor[neighbor + 1];
        int neighbor_now = neighbor_start + threadIdx.x;
        while (neighbor_now < neighbor_end) {
          int temp = adj_list_neighbor[neighbor_now];
          int bin = temp & MODULO;
          P_counter += linear_search(temp, shared_partition, partition,
                                     bin_count, bin + BIN_OFFSET, BIN_START);
          neighbor_now += blockDim.x;
        }
        now++;
      }
    } else {
      int superwarp_ID = threadIdx.x / 64;
      int superwarp_TID = threadIdx.x % 64;
      int workid = superwarp_TID;
      now = now + superwarp_ID;
      while (now < end) {
        int neighbor = adj_list_intersection[now];
        int neighbor_start = beg_pos_neighbor[neighbor];
        int neighbor_end = beg_pos_neighbor[neighbor + 1];
        while (now < end && workid - (neighbor_end - neighbor_start) >= 0) {
          now += 16;
          workid -= (neighbor_end - neighbor_start);
          neighbor = adj_list_intersection[now];
          neighbor_start = beg_pos_neighbor[neighbor];
          neighbor_end = beg_pos_neighbor[neighbor + 1];
        }
        if (now < end) {
          int temp = adj_list_neighbor[neighbor_start + workid];
          int bin = temp & MODULO;
          P_counter += linear_search(temp, shared_partition, partition,
                                     bin_count, bin + BIN_OFFSET, BIN_START);
        }
        workid += 64;
      }
    }

    // unsigned long long intersection_time=clock64()-start_time;
    // if (threadIdx.x==0 &&degree>3000)
    // {
    // 	int max_len_collision= max_count(bin_count,0,blockDim.x,1);
    // 	printf("%d %d %d %d %lld
    // %lld\n",degree,vertexID,blockIdx.x,max_len_collision,hash_time,intersection_time);
    // }

    __syncthreads();
    // if (vertex>1) break;
    if (use_static) {
      vertex += gridDim.x * total_process;
    } else {
      vertex++;
      if (vertex == vertex_end) {
        if (threadIdx.x == 0) {
          ver = atomicAdd(&G_INDEX[1], CHUNK_SIZE * total_process);
        }
        __syncthreads();
        vertex = ver;
        vertex_end = vertex + CHUNK_SIZE;
      }
    }
    __syncthreads();
  }
  __syncthreads();
  // unsigned long long CTA_time=clock64()-start_time;
  // start_time = clock64();

  // warp method
  int WARPID = threadIdx.x / WARPSIZE;
  int WARP_TID = threadIdx.x % WARPSIZE;
  int WARPDIM = blockDim.x * gridDim.x / WARPSIZE;
  vertex =
      warpfirstvertex +
      ((WARPID + blockIdx.x * blockDim.x / WARPSIZE) * total_process + rank) *
          CHUNK_SIZE;
  vertex_end = vertex + CHUNK_SIZE;
  // vertex++;
  // while (vertex<warpfirstvertex+32768*2)
  while (vertex < nothreadfirstvertex) {
    int vertexID = vertexmap[vertex];
    // if (vertex==warpfirstvertex+32768*2-1 && WARP_TID==0)
    // 	printf("%d %d ok\n",vertex,vertexID);
    // unsigned long long start_time=clock64();
    int BINsize = blockDim.x * BUCKET_SIZE;
    int BIN_START = blockIdx.x * BINsize;
    int start = beg_pos_HT[vertexID];
    int end = beg_pos_HT[vertexID + 1];
    int degree = end - start;
    int now = WARP_TID + start;
    int MODULO = WARPSIZE - 1;
    int BIN_OFFSET = WARPID * WARPSIZE;

    // if (vertex==warpfirstvertex+32768*2-1 && WARP_TID==0)
    // 	printf("%d %d\n",start,end);
    // clean bin_count
    // unsigned long long hash_start=clock64();
    bin_count[threadIdx.x] = 0;
    __syncwarp();

    // count hash bin
    while (now < end) {
      int temp = adj_list_HT[now];
      int bin = temp & MODULO;
      bin += BIN_OFFSET;
      int index;
      for (;;) {
        index = atomicAdd(&bin_count[bin], 1);
        if (index < shared_BUCKET_SIZE) {
          shared_partition[index * blockDim.x + bin] = temp;
          break;
        } else if (index < BUCKET_SIZE) {
          index = index - shared_BUCKET_SIZE;
          partition[index * blockDim.x + bin + BIN_START] = temp;
          break;
        }
        break;
        index = atomicAdd(&bin_count[bin], -1);
        bin++;
        if (bin - BIN_OFFSET == 32)
          bin = BIN_OFFSET;
      }
      now += WARPSIZE;
    }
    __syncwarp();

    // if (vertex==warpfirstvertex+32768*2-1 && WARP_TID==0)
    // {
    // 	printf("\n");
    // 	for (int i=0;i<32;i++)
    // 		printf("%d ",bin_count[i+BIN_OFFSET]);
    // }
    // return ;
    // unsigned long long hash_time=clock64()-hash_start;
    // unsigned long long intersection_start=clock64();
    // list intersection
    now = beg_pos_intersection[vertexID];
    end = beg_pos_intersection[vertexID + 1];

    if (without_combination) {
      while (now < end) {
        int neighbor = adj_list_intersection[now];
        int neighbor_start = beg_pos_neighbor[neighbor];
        int neighbor_end = beg_pos_neighbor[neighbor + 1];
        int neighbor_now = neighbor_start + WARP_TID;
        while (neighbor_now < neighbor_end) {
          int temp = adj_list_neighbor[neighbor_now];
          int bin = temp & MODULO;
          P_counter += linear_search(temp, shared_partition, partition,
                                     bin_count, bin + BIN_OFFSET, BIN_START);
          neighbor_now += WARPSIZE;
        }
        now++;
      }
    } else {
      int workid = WARP_TID;
      while (now < end) {
        int neighbor = adj_list_intersection[now];
        int neighbor_start = beg_pos_neighbor[neighbor];
        int neighbor_end = beg_pos_neighbor[neighbor + 1];
        while (now < end && workid - (neighbor_end - neighbor_start) >= 0) {
          now++;
          workid -= (neighbor_end - neighbor_start);
          neighbor = adj_list_intersection[now];
          neighbor_start = beg_pos_neighbor[neighbor];
          neighbor_end = beg_pos_neighbor[neighbor + 1];
        }
        __syncwarp();
        if (now < end) {
          int temp = adj_list_neighbor[neighbor_start + workid];
          int bin = temp & MODULO;
          P_counter += linear_search(temp, shared_partition, partition,
                                     bin_count, bin + BIN_OFFSET, BIN_START);
        }
        __syncwarp();
        now = __shfl_sync(0xffffffff, now, 31);
        workid = __shfl_sync(0xffffffff, workid, 31);

        workid += WARP_TID + 1;

        // workid+=WARPSIZE;
      }
    }
    __syncwarp();
    // unsigned long long intersection_time=clock64()-intersection_start;
    // unsigned long long total_time=clock64()-start_time;
    // if(threadIdx.x%32==0){
    // 	// printf("%d %d %d\n",total_time, hash_time, intersection_time);
    // 	// TT+=total_time;
    // 	// HT+=hash_time;
    // 	// IT+=intersection_time;
    // 	gettime[vertexID]=total_time;
    // 	maxcollision[vertex]=max_count(bin_count,BIN_OFFSET,BIN_OFFSET+WARPSIZE,0);
    // }
    // if(threadIdx.x%32==0){
    // 	gettime[vertex]=1;}
    // __syncwarp();
    // if (vertex>1) break;
    if (use_static) {
      vertex += WARPDIM * total_process;
    } else {
      vertex++;
      if (vertex == vertex_end) {
        if (WARP_TID == 0) {
          vertex = atomicAdd(&G_INDEX[2], CHUNK_SIZE * total_process);
        }
        __syncwarp();
        vertex = __shfl_sync(0xffffffff, vertex, 0);
        vertex_end = vertex + CHUNK_SIZE;
      }
    }
  }

  // unsigned long long warp_time=clock64()-start_time;

  // if (threadIdx.x==0)
  // {
  // 	printf("%d %lld %lld\n",blockIdx.x,CTA_time,warp_time);
  // }
  atomicAdd(&G_counter, P_counter);
  // atomicAdd(&G_HT,HT);
  // atomicAdd(&G_TT,TT);
  // atomicAdd(&G_IT,IT);

  __syncthreads();
  if (threadIdx.x == 0) {
    // printf("%d\n",G_TT);
    atomicAdd(&GLOBAL_COUNT[0], G_counter);
    // atomicAdd(&GLOBAL_COUNT[1],G_TT);
    // atomicAdd(&GLOBAL_COUNT[2],G_HT);
    // atomicAdd(&GLOBAL_COUNT[3],G_IT);
  }
}

struct arguments Triangle_count(int rank, char name[100], struct arguments args,
                                int total_process, int n_threads, int n_blocks,
                                int chunk_size, int partition_num) {

  // int partition_num=2;// should be n
  int T_Group = 32;
  int PER_BLOCK_WARP = n_threads / T_Group;
  int total = n_blocks * PER_BLOCK_WARP * 32 * BUCKET_SIZE;
  unsigned long long *counter =
      (unsigned long long *)malloc(sizeof(unsigned long long) * 10);
  string json_file = name;
  stringstream ii;
  stringstream jj;
  stringstream kk;
  ii << rank % partition_num;
  jj << rank / partition_num % partition_num;

  kk << rank / partition_num / partition_num % partition_num;
  graph *graph_HT =
      new graph(json_file + "/partition" + ii.str() + "_" + jj.str());
  graph *graph_intersection =
      new graph(json_file + "/partition" + ii.str() + "_" + kk.str());
  graph *graph_neighbor =
      new graph(json_file + "/partition" + kk.str() + "_" + jj.str());

  json_file = json_file + "/partition" + ii.str() + "_" + kk.str();
  string loadfile = json_file + "/division";
  fstream inFile(loadfile.c_str(), ios::in);
  int warpfirstvertex, nothreadfirstvertex;
  inFile >> warpfirstvertex >> nothreadfirstvertex;
  // cout<<warpfirstvertex<<' '<<nothreadfirstvertex<<endl;
  // warpfirstvertex=nothreadfirstvertex;
  // warpfirstvertex--;
  // nothreadfirstvertex++;
  // warpfirstvertex=0;
  inFile.close();
  loadfile = json_file + "/map";

  inFile.open(loadfile.c_str(), ios::in | ios::binary);
  int *vertexmap = new int[nothreadfirstvertex];
  inFile.read((char *)vertexmap, nothreadfirstvertex * sizeof(int));
  inFile.close();

  int deviceCount;
  HRR(hipGetDeviceCount(&deviceCount));
  HRR(hipSetDevice(rank % deviceCount));

  // float memory_req = (sizeof(int)*total + sizeof(index_t)*(vertex_count+1)+
  // sizeof(vertex_t)*(edge_count)+sizeof(vertex_t)*(edge_list_count))/(1024*1024);
  // fprintf(stderr,"-------------------GPU: %d, Memory required: %f
  // MB\n",rank,memory_req);
  // printf("%f\n",memory_req);
  rank = rank / partition_num / partition_num / partition_num;
  total_process = total_process / partition_num / partition_num / partition_num;
  int *hash, *BIN_MEM;
  unsigned long long *GLOBAL_COUNT, *g_gettime, *g_maxcollision;
  int *G_INDEX, *g_vertexmap;

  // cout<<graph_HT-> vert_count<<' '<<graph_HT-> edge_count<<endl;
  index_t vertex_count = graph_HT->vert_count;
  index_t edge_count = graph_HT->edge_count;
  index_t edge_list_count = graph_HT->edge_list_count;
  index_t edges = graph_HT->edge_count;

  index_t *d_beg_pos_HT;
  vertex_t *d_adj_list_HT;

  // cout<<vertex_count<<' '<<edge_count<<endl;
  HRR(hipMalloc((void **)&d_beg_pos_HT, sizeof(index_t) * (vertex_count + 1)));
  HRR(hipMalloc((void **)&d_adj_list_HT, sizeof(vertex_t) * (edge_count)));

  HRR(hipMemcpy(d_beg_pos_HT, graph_HT->beg_pos,
                 sizeof(index_t) * (vertex_count + 1), hipMemcpyHostToDevice));
  HRR(hipMemcpy(d_adj_list_HT, graph_HT->adj_list,
                 sizeof(vertex_t) * edge_count, hipMemcpyHostToDevice));

  vertex_count = graph_intersection->vert_count;
  edge_count = graph_intersection->edge_count;
  edge_list_count = graph_intersection->edge_list_count;
  edges = graph_intersection->edge_count;

  index_t *d_beg_pos_intersection;
  vertex_t *d_adj_list_intersection;

  HRR(hipMalloc((void **)&d_beg_pos_intersection,
                 sizeof(index_t) * (vertex_count + 1)));
  HRR(hipMalloc((void **)&d_adj_list_intersection,
                 sizeof(vertex_t) * (edge_count)));

  HRR(hipMemcpy(d_beg_pos_intersection, graph_intersection->beg_pos,
                 sizeof(index_t) * (vertex_count + 1), hipMemcpyHostToDevice));
  HRR(hipMemcpy(d_adj_list_intersection, graph_intersection->adj_list,
                 sizeof(vertex_t) * edge_count, hipMemcpyHostToDevice));

  vertex_count = graph_neighbor->vert_count;
  edge_count = graph_neighbor->edge_count;
  edge_list_count = graph_neighbor->edge_list_count;
  edges = graph_neighbor->edge_count;

  index_t *d_beg_pos_neighbor;
  vertex_t *d_adj_list_neighbor;

  HRR(hipMalloc((void **)&d_beg_pos_neighbor,
                 sizeof(index_t) * (vertex_count + 1)));
  HRR(hipMalloc((void **)&d_adj_list_neighbor,
                 sizeof(vertex_t) * (edge_count)));

  HRR(hipMemcpy(d_beg_pos_neighbor, graph_neighbor->beg_pos,
                 sizeof(index_t) * (vertex_count + 1), hipMemcpyHostToDevice));
  HRR(hipMemcpy(d_adj_list_neighbor, graph_neighbor->adj_list,
                 sizeof(vertex_t) * edge_count, hipMemcpyHostToDevice));
  // cout<<"total:"<<total<<endl;

  if (1) {
    HRR(hipMalloc((void **)&GLOBAL_COUNT, sizeof(unsigned long long) * 10));
    HRR(hipMalloc((void **)&g_gettime,
                   sizeof(unsigned long long) * (vertex_count + 1)));
    HRR(hipMalloc((void **)&g_maxcollision,
                   sizeof(unsigned long long) * (vertex_count + 1)));
    HRR(hipMalloc((void **)&G_INDEX, sizeof(int) * 3));
    HRR(hipMalloc((void **)&BIN_MEM, sizeof(int) * total));
    HRR(hipMalloc((void **)&g_vertexmap, sizeof(int) * (nothreadfirstvertex)));

    int nowindex[3];
    nowindex[0] = chunk_size * n_blocks * n_threads / T_Group;
    nowindex[1] = chunk_size * (n_blocks * total_process + rank);
    nowindex[2] =
        warpfirstvertex +
        chunk_size * (n_blocks * n_threads / T_Group * total_process + rank);
    // unsigned long long cou=0;
    // int nowindex=0;

    HRR(hipMemcpy(G_INDEX, nowindex, sizeof(int) * 3, hipMemcpyHostToDevice));
    HRR(hipMemcpy(g_vertexmap, vertexmap, sizeof(int) * (nothreadfirstvertex),
                   hipMemcpyHostToDevice));
  }

  double t1 = wtime();
  double cmp_time;

  if (1) {
    double time_start = wtime();
    dynamic_assign<<<n_blocks, n_threads>>>(
        d_adj_list_HT, d_beg_pos_HT, d_adj_list_intersection,
        d_beg_pos_intersection, d_adj_list_neighbor, d_beg_pos_neighbor,
        edge_count, vertex_count, edge_list_count, BIN_MEM, GLOBAL_COUNT, rank,
        total_process, BUCKET_SIZE, T_Group, G_INDEX, chunk_size,
        warpfirstvertex, nothreadfirstvertex, g_vertexmap, g_gettime,
        g_maxcollision);
    HRR(hipDeviceSynchronize());
    cmp_time = wtime() - time_start;
  }

  HRR(hipMemcpy(counter, GLOBAL_COUNT, sizeof(unsigned long long) * 10,
                 hipMemcpyDeviceToHost));
  // unsigned long long *gettime= new unsigned long long[vertex_count+1];
  // HRR(hipMemcpy(gettime,g_gettime,sizeof(unsigned long
  // long)*(vertex_count+1), hipMemcpyDeviceToHost)); unsigned long long
  // *maxcollision= new unsigned long long[vertex_count+1];
  // HRR(hipMemcpy(maxcollision,g_maxcollision,sizeof(unsigned long
  // long)*(vertex_count+1), hipMemcpyDeviceToHost));

  HRR(hipFree(GLOBAL_COUNT));
  HRR(hipFree(g_gettime));
  HRR(hipFree(g_maxcollision));
  HRR(hipFree(G_INDEX));
  HRR(hipFree(BIN_MEM));
  HRR(hipFree(g_vertexmap));
  HRR(hipFree(d_beg_pos_HT));
  HRR(hipFree(d_adj_list_HT));
  HRR(hipFree(d_beg_pos_intersection));
  HRR(hipFree(d_adj_list_intersection));
  HRR(hipFree(d_beg_pos_neighbor));
  HRR(hipFree(d_adj_list_neighbor));
  // free(counter);

  delete graph_HT;
  delete graph_intersection;
  delete graph_neighbor;
  delete vertexmap;
  args.time = cmp_time;
  args.count = counter[0];
  // cout<<counter[0]<<endl;
  // printf("%lld\n",args.count);
  // cout<<counter[1]<<' '<<counter[2]<<' '<<counter[3]<<endl;
  // for (int i=0;i<vertex_count;i++)
  // 	if (gettime[i]>0)
  // 		cout<<gettime[i]<<'
  // '<<graph_d->beg_pos[i+1]-graph_d->beg_pos[i]<<'
  // '<<maxcollision[i]+1<<endl;

  // for (int i=0;i<vertex_count;i++)
  // 	if (gettime[i]==0&&i%total_process==rank ||
  // gettime[i]==1&&i%total_process!=rank)
  // 	{
  // 		cout<<i<<endl;
  // 		break;
  // 	}
  args.edge_count = edges;
  // args.degree= SIZE;
  args.vertices = vertex_count - 1;
  return args;
}
